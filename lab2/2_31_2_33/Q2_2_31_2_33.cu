//Norhan_Reda_Abdelwahed_2_31
//Hoda_Gamal_Hamouda_2_33

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <iostream>
#include <fstream>
#include <vector>
using namespace std;
#define N 800
// Kernel definition
__global__ void MatMul(float *A, float *B, float *C, int rows, int cols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < rows) {
        float element = 0.0f;
        for (int col = 0; col < cols; col++) {
            element += A[row * cols + col] * B[col];
        }
        C[row] = element;
    }

}

int main(int argc, char *argv[])
{


    // Specify the file path
    std::string filePath = argv[1];

    // Open the file for reading
    std::ifstream inputFile(filePath);
    std::string outputFilePath = argv[2];
    // Check if the file is opened successfully
    if (!inputFile.is_open()) {
        std::cerr << "Error opening file: " << filePath << std::endl;
        return 1; // Exit with an error code
    }

    // Open the output file for writing
    std::ofstream outputFile(outputFilePath);
      // Check if the output file is opened successfully
    if (!outputFile.is_open()) {
        std::cerr << "Error opening output file: " << outputFilePath << std::endl;
        inputFile.close(); // Close the input file before exiting
        return 1; // Exit with an error code
    }


    // Read the number of test cases
    int numTestCases;
    int rows ;
    int cols;
    inputFile >> numTestCases;

    // Process each test case
    for (int testCase = 1; testCase <= numTestCases; ++testCase) {
        std::cout << "Test Case " << testCase << ":" << std::endl;

        // Read the number of rows and columns for the current test case

        inputFile >> rows >> cols;

        cout<<rows<<","<<cols<<endl;
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    // Allocate host memory
    h_A = (float*)malloc(sizeof(float) * rows * cols);
    h_B =(float*)malloc(sizeof(float) * cols *1 );
    h_C = (float*)malloc(sizeof(float) * rows * 1);

    // Populate matrices h_A and h_B

    // Allocate device memory
    hipMalloc((void**)&d_A, rows * cols * sizeof(float));
    hipMalloc((void**)&d_B, cols * 1 * sizeof(float));
    hipMalloc((void**)&d_C, rows * 1 * sizeof(float));




        // Read matrix1 elements
        for (int i = 0; i < rows; ++i) {
            for (int j = 0; j < cols; ++j) {
                inputFile >> h_A[i * cols + j];
            }
        }

        // Read matrix2 elements
        for (int i = 0; i < cols; ++i) {

                inputFile >> h_B[i ];

        }

        // Print matrices for the current test case (you can replace this with your specific logic)
        std::cout << "Matrix1:" << std::endl;
        for (int i = 0; i < rows; ++i) {
            for (int j = 0; j < cols; ++j) {
                std::cout << h_A[i * cols + j] << " ";
            }
            std::cout << std::endl;
        }

        std::cout << "Matrix2:" << std::endl;
        for (int i = 0; i < cols; ++i) {
            for (int j = 0; j < 1; ++j) {
                std::cout << h_B[i * cols + j] << " ";
            }
            std::cout << std::endl;
        }

      // Copy data from host to device
    hipMemcpy(d_A, h_A, rows * cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, cols * 1 * sizeof(float), hipMemcpyHostToDevice);



    //...
    // Kernel invocation with one block of N * N * 1 threads
    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((cols-1) / threadsPerBlock.x+1,(rows-1) / threadsPerBlock.y+1);
    MatMul<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C,rows,cols);




    //...
    // Copy the result matrix C from device to host
    hipMemcpy(h_C, d_C, rows * 1 * sizeof(float), hipMemcpyDeviceToHost);

    printf("h_C[0] = %f\n", h_C[0]);
    printf("PASSED\n");
    // Write matrices for the current test case to the output file

       for (int i = 0; i < rows; ++i) {

                outputFile << h_C[i]<<" ";

            outputFile << std::endl;
        }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    }

    // Close the file
    inputFile.close();
    outputFile.close();

   return 0;
}
