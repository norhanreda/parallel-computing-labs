//Norhan_Reda_Abdelwahed_2_31
//Hoda_Gamal_Hamouda_2_33


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <vector>

using namespace std;

// Kernel definition
__global__ void ArrAdd(double *arr, long size,double *result)
{
    extern __shared__ double partialSum[];
    unsigned int t = threadIdx.x;
    
    unsigned int new_size = (size-1) / blockDim.x +1 ;
    partialSum[t] = 0;

    for(int i=0;i<new_size ;i++)
    {
      partialSum[t]+= arr[t*new_size+i];
    }

    if(t >=size)
      return ;
    for(unsigned int stride = 1; stride < blockDim.x; stride *= 2){

        if(t % (2*stride) == 0)
              partialSum[t] += partialSum[t+stride];
        __syncthreads();
    }

    if(t==0)
        *result = partialSum[0];
    }

int main(int argc, char *argv[])
{

    // Specify the file path
    std::string filePath = argv[1];

    // Open the file for reading
    std::ifstream file(filePath);

   if (!file.is_open()) {
        std::cerr << "Error opening file" << std::endl;
        return 1;
    }

    double* array = nullptr;  // Pointer to the array
    double * result = nullptr;
    double value;
    long count = 0;  // Variable to keep track of the number of elements

    // Read elements from the file
    while (file >> value) {
        // Dynamically resize the array
        double* temp = static_cast<double*>(realloc(array, (count + 1) * sizeof(double)));

        if (temp == nullptr) {
            std::cerr << "Error allocating memory" << std::endl;
            free(array);  // Free the previously allocated memory
            return 1;
        }

        array = temp;

        // Add the read value to the array
        array[count] = value;

        count++;
    }

    // Close the file
    file.close();

    // host to device allocation
     double *array_device;
     double *result_device;

     result = (double*)malloc(sizeof(double));
     hipMalloc((void**)&array_device, count * sizeof(long));
     hipMalloc((void**)&result_device,sizeof(double));

    hipMemcpy(array_device, array, count * sizeof(long), hipMemcpyHostToDevice);
    dim3 threadsPerBlock(1024, 1);
    dim3 numBlocks(1, 1);
    ArrAdd<<< numBlocks, threadsPerBlock , threadsPerBlock.x * sizeof(double) >>>(array_device,count,result_device);

    hipMemcpy( result, result_device, sizeof(double), hipMemcpyDeviceToHost);

    //cout<<*result<<endl;
    cout << fixed <<setprecision(2) << *result <<endl;

    // Free the dynamically allocated memory in host
    free(array);
    free(result);

    // Free device memory
    hipFree(array_device);
    hipFree(result_device);

   return 0;
}
