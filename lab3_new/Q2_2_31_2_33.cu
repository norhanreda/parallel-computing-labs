//Norhan_Reda_Abdelwahed_2_31
//Hoda_Gamal_Hamouda_2_33


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <iostream>
#include <fstream>
#include <vector>

using namespace std;

// Kernel definition
__global__ void BinarySearch(double *array, long size, double target, long *index)
{

    __shared__ long shared_start;
    __shared__ long shared_end;
    __shared__ bool shared_found;
    __shared__ bool target_found;

    int tid = threadIdx.x;

    *index = -1;

    if (tid == 0) {
        shared_start = 0;
        shared_end = size - 1;
        shared_found = false;
        target_found = false;
    }
    __syncthreads();

    while (shared_start <= shared_end && shared_found == false && target_found == false) {

        long elements_per_thread = (shared_end - shared_start + 1 + blockDim.x - 1) / blockDim.x;
        long start = min(shared_start + tid * elements_per_thread, shared_end);
        long end = min(start + elements_per_thread - 1, shared_end);

        /*
        if (end > shared_end) {
            end = shared_end;
        }
        if(start > end){
          start = end;
        }*/


        if (array[start] <= target && target <= array[end]) {
            shared_start = start;
            shared_end = end;
            shared_found = true;
        }

        __syncthreads();

        if (shared_found == false && shared_start == shared_end){
          target_found = true;
          *index = -1;
        }
        else if (shared_found == true && shared_start == shared_end){
          target_found = true;
          *index = shared_start;
        }
        else if(shared_found == false){
           target_found = true;
           *index = -1;
        }
        shared_found = false;

    }
}


int main(int argc, char *argv[])
{


    // Specify the file path
    std::string filePath = argv[1];
    double target = atof(argv[2]);

    // Open the file for reading
    std::ifstream file(filePath);

   if (!file.is_open()) {
        std::cerr << "Error opening file" << std::endl;
        return 1;
    }

    double* array = nullptr;  // Pointer to the array
    long * result = nullptr;
    double value;
    long count = 0;  // Variable to keep track of the number of elements

    // Read elements from the file
    while (file >> value) {
        // Dynamically resize the array
        double* temp = static_cast<double*>(realloc(array, (count + 1) * sizeof(double)));

        if (temp == nullptr) {
            std::cerr << "Error allocating memory" << std::endl;
            free(array);  // Free the previously allocated memory
            return 1;
        }

        array = temp;

        // Add the read value to the array
        array[count] = value;

        count++;
    }

    // Close the file
    file.close();


     // host to device allocation
     double *array_device;
     long *result_device;

     result = (long*)malloc(sizeof(long));

     hipMalloc((void**)&array_device, count * sizeof(long));
     hipMalloc((void**)&result_device,sizeof(long));

    hipMemcpy(array_device, array, count * sizeof(double), hipMemcpyHostToDevice);


    dim3 threadsPerBlock(1024, 1);
    dim3 numBlocks(1,1);

    BinarySearch<<< numBlocks, threadsPerBlock>>>(array_device,count, target,result_device);

    hipMemcpy( result, result_device, sizeof(long), hipMemcpyDeviceToHost);

    cout<<*result<<endl;


    // Free the dynamically allocated memory in host
    free(array);
    free(result);

    // Free device memory
    hipFree(array_device);
    hipFree(result_device);

   return 0;
}
