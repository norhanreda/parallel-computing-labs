#include "hip/hip_runtime.h"
//Norhan_Reda_Abdelwahed_2_31
//Hoda_Gamal_Hamouda_2_33

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <vector>
#define STB_IMAGE_IMPLEMENTATION
#include <stdio.h>
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include <string>
#include <filesystem>
#include <hip/hip_runtime.h>

using namespace std;
__constant__ float mask[15*15];


__global__ void Imgcov3d(int batchSize, float *input, float* output, unsigned int width,unsigned int height,int maskdim, int shared_width,int shared_height, int shared_depth){

  extern __shared__ float sharedTile [];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int tile_x = shared_width - (maskdim - 1);
    int tile_y = shared_height - (maskdim - 1);

    int x_o = blockIdx.x * tile_x + threadIdx.x;
    int y_o = blockIdx.y * tile_y + threadIdx.y;
    int idx = blockIdx.z * blockDim.z + threadIdx.z;

    if (idx >= batchSize){
      return;
    }

    // number of pixels each thread should get from input image
    int num = ((shared_width*shared_height) - 1) /(tile_x*tile_y) + 1;

    // each thread gets 'num' pixel (its 3 channels) from input image
    for (int d=0; d< 3; d++){
      for(int f=0; f < num ; f++){

          int element_index = tile_x * tile_y * f + ty * tile_x + tx;

          // get col, row in shared memory
          int dx = element_index % shared_width;
          element_index = element_index / shared_width;
          int dy = element_index % shared_height;

          // get col, row in input image
          int x_i = dx + blockIdx.x * tile_x - maskdim/2;
          int y_i = dy + blockIdx.y * tile_y - maskdim/2;

          if(x_i >= 0 && y_i >= 0 && x_i < width && y_i < height){
                sharedTile[d * shared_height * shared_width + dy * shared_width + dx]
                = input[idx * width * height * 3 + d * height * width + y_i * width + x_i];
          }
          else{
              sharedTile[d * shared_height * shared_width + dy * shared_width + dx] = 0.0;
          }
      }
    }

    __syncthreads();


  // all threads contribute in calculating the output
  if(y_o < height && x_o < width){

  float sum = 0;

    for(int y_mask = 0; y_mask < maskdim; y_mask++) {
            for(int x_mask = 0; x_mask < maskdim; x_mask++) {
                  for(int z_mask = 0; z_mask < 3; z_mask++) {
                    sum += mask[y_mask * maskdim + x_mask] *
                    sharedTile[z_mask * shared_height * shared_width + (ty + y_mask) * shared_width +(tx + x_mask)];

                }
            }
        }

    if(sum < 0){
      sum = 0;
    } else if (sum > 255){
      sum = 255;
    }

        output[idx * width * height + y_o * width + x_o] = sum;
    }
}

int main(int argc, char *argv[]) {

    string folderPath = argv[1];
    string out_path = argv[2];
    int batch_size = stoi(argv[3]);
    string mask_path = argv[4];

    int maskdim;

    // Open the file for reading
    std::ifstream inputFile(mask_path);

    // Check if the file is opened successfully
    if (!inputFile.is_open()) {
        std::cerr << "Error opening file: " <<mask_path  << std::endl;
        return 1; // Exit with an error code
    }

    inputFile >> maskdim ;
     //cout<<"maskdim "<<maskdim<<endl;

    float *h_A, *h_B,*R,*G,*B;
    int rows1,cols1, comp;

    float *h_images;
    int img_index = 0;

    for (const auto& entry : std::filesystem::directory_iterator(folderPath)) {

        if (entry.is_regular_file()) {
            std::string filePath = entry.path().string();

            unsigned char *data = stbi_load(filePath.c_str(), &cols1, &rows1, &comp, 0);

            if(img_index == 0){
              h_images = (float*)malloc(sizeof(float) * batch_size * cols1 * rows1 * comp);
            }

            if (data) {

                /*printf("cols %d\n", cols1);
                printf("rows %d\n", rows1);
                printf("image %s\n", filePath.c_str());
                */

                h_A = (float*)malloc(sizeof(float) *  rows1 * cols1* comp);
                R = (float*)malloc(sizeof(float) *  rows1 * cols1);
                G = (float*)malloc(sizeof(float) *  rows1 * cols1);
                B = (float*)malloc(sizeof(float) *  rows1 * cols1);

                int k = 0;
                for(int i=0;i<rows1 * cols1* comp;i=i+3)
                {
                  R[k] =  static_cast<float>(data[i]);
                  k=k+1;
                }
                k=0;
                for(int i=1;i<rows1 * cols1* comp;i=i+3)
                {
                  G[k] =  static_cast<float>(data[i]);
                  k=k+1;
                }
                k=0;
                for(int i=2;i<rows1 * cols1* comp;i=i+3)
                {
                  B[k] =  static_cast<float>(data[i]);
                  k=k+1;
                }


                memcpy(h_A, R, rows1*cols1 * sizeof(float));
                memcpy(h_A +rows1*cols1 , G, rows1*cols1 * sizeof(float));
                memcpy(h_A+rows1*cols1+rows1*cols1, B, rows1*cols1 * sizeof(float));

                memcpy(h_images + img_index * rows1 * cols1 * 3,                     R, rows1 * cols1 * sizeof(float));
                memcpy(h_images + img_index * rows1 * cols1 * 3 + rows1 * cols1,     G, rows1 * cols1 * sizeof(float));
                memcpy(h_images + img_index * rows1 * cols1 * 3 + rows1 * cols1 * 2, B, rows1 * cols1 * sizeof(float));

                img_index++;

             }
             else {
                // Failed to load the image
                std::cerr << "Failed to load image: " << filePath << std::endl;
            }

           }
      }


      float *d_images_temp = (float*)malloc(sizeof(float) * batch_size * rows1 * cols1 * comp);
      for(int i=0; i<batch_size; i++){
        memcpy(d_images_temp + i * rows1 * cols1 *comp , h_images + i * rows1 * cols1 *comp, rows1 * cols1 *comp * sizeof(float));
      }


    float *d_images;
    hipMalloc((void**)&d_images, batch_size * rows1 * cols1 * comp * sizeof(float));
    hipMemcpy(d_images, d_images_temp, batch_size * rows1 * cols1 * comp * sizeof(float), hipMemcpyHostToDevice);

    float *d_outs;
    hipMalloc((void**)&d_outs, batch_size * rows1 * cols1 * sizeof(float));

    float *h_outs = (float*)malloc(sizeof(float) * batch_size * rows1 * cols1);

    h_B =(float*)malloc(sizeof(float) * maskdim* maskdim);

    // Read mask elements
    for (int i = 0; i < maskdim; ++i) {
        for (int j = 0; j < maskdim; ++j) {

            int index =  i * maskdim + j;
            inputFile >>  h_B[index];

        }
    }

    // Copy data from host to device
    hipMemcpyToSymbol(HIP_SYMBOL(mask), h_B, maskdim * maskdim * sizeof(float), 0, hipMemcpyHostToDevice);


    // Kernel invocation

    dim3 threadsPerBlock(16,16,1);

    int tile_x = threadsPerBlock.x;
    int tile_y = threadsPerBlock.y;

    int x_blocks = (cols1 + tile_x - 1) / tile_x;
    int y_blocks = (rows1 + tile_y - 1) / tile_y;
    int z_blocks = batch_size;

    int shared_width = tile_x + maskdim - 1;
    int shared_height = tile_y + maskdim - 1;
    int shared_depth = threadsPerBlock.z;

    dim3 numBlocks(x_blocks, y_blocks, z_blocks);

    Imgcov3d<<<numBlocks, threadsPerBlock,(sizeof(float)*shared_width*shared_height*3*shared_depth)>>>(batch_size,d_images,d_outs,cols1,rows1, maskdim,shared_width,shared_height,shared_depth);

    hipMemcpy(h_outs, d_outs, batch_size * rows1 * cols1 * sizeof(float), hipMemcpyDeviceToHost);

    unsigned char* h_char = (unsigned char*)malloc(sizeof(unsigned char) * rows1 * cols1);

    for(int k=0;k<batch_size;k++){

      for(int i=0;i<rows1*cols1;i++){
          h_char[i]=static_cast<unsigned char>(h_outs[k*rows1*cols1+i]);
      }
      string filename = "./"+out_path+"/"+"image" + std::to_string(k) + ".jpg";
      stbi_write_jpg(filename.c_str(), cols1, rows1, 1, h_char, 100);
    }

    // Free device memory
    hipFree(d_images);
    hipFree(d_outs);

    // Free host memory
    free(h_A);
    free(h_B);
    free(R);
    free(G);
    free(B);
    free(h_images);
    free(d_images_temp);
    free(h_outs);
    free(h_char);

  return 0;
}
